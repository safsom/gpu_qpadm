/*
 * precompute_gpu.cu - GPU precomputation of Gram matrices for qpAdm
 *
 * This module constructs blockwise population‑wise Gram matrices
 * G_b = X_b^T X_b from the per‑population allele frequency matrix
 * X_b (m_b x P).  Each Gram matrix is symmetric of size P x P and
 * accumulates the contributions of m_b SNPs in a jackknife block.
 *
 * The heavy lifting is offloaded to the GPU via cuBLAS DGEMM.  We
 * allocate device memory for each block, copy the column‑major
 * genotype means, perform a single DGEMM call and copy the result
 * back to the host.  All Gram matrices are appended to a binary
 * cache file along with a simple header encoding the dimensions.
 *
 * Note: error handling is minimal—production code should check
 * return codes more carefully and handle allocation failures.  This
 * implementation assumes the GPU has enough memory to handle
 * reasonably sized blocks (m_b × P).  If blocks are extremely
 * large, consider streaming the computation in tiles.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

/* bring in GenoData definition from io.c */
extern "C" {
    typedef struct GenoData GenoData;
    struct GenoData {
        int num_snps;
        int num_inds;
        int num_pops;
        int num_blocks;
        int *block_start;
        int *block_end;
        double **snp_means;
    };
}

/* simple header for the gram cache file */
typedef struct {
    int num_pops;
    int num_blocks;
} GramHeader;

/* CUDA error checking macro */
#define CHECK_CUDA(call) do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        return -1; \
    } \
} while (0)

/* cuBLAS error checking macro */
#define CHECK_CUBLAS(call) do { \
    hipblasStatus_t stat = (call); \
    if (stat != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error %d at %s:%d\n", (int)stat, __FILE__, __LINE__); \
        return -1; \
    } \
} while (0)

/*
 * Compute a single Gram matrix for block b and write it to the file
 * descriptor.  Returns 0 on success or -1 on error.
 */
static int process_block_gpu(const GenoData *gd, int b, FILE *fp, hipblasHandle_t handle) {
    int start = gd->block_start[b];
    int end   = gd->block_end[b];
    int m = end - start;
    int P = gd->num_pops;
    /* allocate host column‑major matrix X (m x P) */
    double *hX = (double *)malloc(sizeof(double) * m * P);
    if (!hX) {
        fprintf(stderr, "Failed to allocate host X for block %d\n", b);
        return -1;
    }
    /* fill X such that X(row, col) = snp_means[col][start + row] */
    for (int col = 0; col < P; col++) {
        for (int row = 0; row < m; row++) {
            hX[row + col * m] = gd->snp_means[col][start + row];
        }
    }
    /* allocate device memory */
    double *dX = NULL;
    double *dG = NULL;
    CHECK_CUDA(hipMalloc((void **)&dX, sizeof(double) * m * P));
    CHECK_CUDA(hipMalloc((void **)&dG, sizeof(double) * P * P));
    /* copy to device */
    CHECK_CUDA(hipMemcpy(dX, hX, sizeof(double) * m * P, hipMemcpyHostToDevice));
    /* compute G = X^T * X using cuBLAS (G and X are column‑major) */
    const double alpha = 1.0;
    const double beta  = 0.0;
    /* A = X^T has dims P x m, B = X has dims m x P, C = G dims P x P */
    CHECK_CUBLAS(hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                             P, P, m,
                             &alpha,
                             dX, m,
                             dX, m,
                             &beta,
                             dG, P));
    /* copy back to host */
    double *hG = (double *)malloc(sizeof(double) * P * P);
    if (!hG) {
        fprintf(stderr, "Failed to allocate host G for block %d\n", b);
        hipFree(dX);
        hipFree(dG);
        free(hX);
        return -1;
    }
    CHECK_CUDA(hipMemcpy(hG, dG, sizeof(double) * P * P, hipMemcpyDeviceToHost));
    /* write G to file in binary, preserving column-major order */
    size_t written = fwrite(hG, sizeof(double), P * P, fp);
    if (written != (size_t)(P * P)) {
        fprintf(stderr, "Failed to write Gram matrix for block %d\n", b);
        hipFree(dX);
        hipFree(dG);
        free(hX);
        free(hG);
        return -1;
    }
    /* cleanup */
    hipFree(dX);
    hipFree(dG);
    free(hX);
    free(hG);
    return 0;
}

/*
 * Public API: precompute all Gram matrices for a given GenoData and
 * write them to a cache file.  The file is created/truncated and
 * contains a header followed by num_blocks consecutive Gram matrices.
 * Each Gram matrix is stored in column-major order as doubles.
 * Returns 0 on success or -1 on failure.
 */
int precompute_gram_gpu(const GenoData *gd, const char *cache_filename) {
    if (!gd || !cache_filename) {
        return -1;
    }
    FILE *fp = fopen(cache_filename, "wb");
    if (!fp) {
        fprintf(stderr, "Failed to open cache file %s for writing\n", cache_filename);
        return -1;
    }
    /* write header */
    GramHeader hdr;
    hdr.num_pops = gd->num_pops;
    hdr.num_blocks = gd->num_blocks;
    if (fwrite(&hdr, sizeof(GramHeader), 1, fp) != 1) {
        fprintf(stderr, "Failed to write header to %s\n", cache_filename);
        fclose(fp);
        return -1;
    }
    /* set up cuBLAS */
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    /* iterate over blocks */
    for (int b = 0; b < gd->num_blocks; b++) {
        if (process_block_gpu(gd, b, fp, handle) != 0) {
            fprintf(stderr, "Error processing block %d\n", b);
            hipblasDestroy(handle);
            fclose(fp);
            return -1;
        }
    }
    hipblasDestroy(handle);
    fclose(fp);
    return 0;
}

/* Function declaration for external linkage */
int precompute_gram_gpu(const GenoData *gd, const char *cache_filename);